#include "hackrf_gpu_gang.hpp"
#include "int8_to_complex_float.hpp"
#include <thrust/complex.h>
using namespace std;

#define BUFLEN 262144
HackRfGpuGang::HackRfGpuGang(vector<int> deviceIds){
	for(auto i:deviceIds){
		m_hackRfs.push_back(new HackRfGpu(i));
	}
	hipMalloc(&computeBuf,BUFLEN*sizeof(thrust::complex<float>)); 
}

HackRfGpuGang::~HackRfGpuGang(){
	for(auto hrf:m_hackRfs){
		delete hrf;
	}
}

void HackRfGpuGang::start(){
	for(auto hrf:m_hackRfs){
		hrf->start();
	}
}

void HackRfGpuGang::process(){
	float *tbuf;
	Int8ToComplexFloat proc;
	//int8_t buf;
	for(auto hrf:m_hackRfs){
		tbuf=hrf->m_itb->consumerCheckout();
		if(tbuf != 0){

			//hipMemcpy(buf, tbuf, BUFLEN, hipMemcpyDeviceToHost;
			proc.process((int8_t*)tbuf, BUFLEN, (thrust::complex<float> *)computeBuf, BUFLEN*sizeof(thrust::complex<float>));
			hrf->m_itb->consumerCheckin();
		} else {
			cout << "Got null, itb->isStopped: " << hrf->m_itb->isStopped();
		}
	}
}

void HackRfGpuGang::stop(){
	for(auto hrf:m_hackRfs){
		hrf->stop();
	}
}
