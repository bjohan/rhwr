#include "hackrf_gpu_gang.hpp"
#include "int8_to_complex_float.hpp"
#include <thrust/complex.h>
#include <stdexcept>
using namespace std;

#define BUFLEN 262144
HackRfGpuGang::HackRfGpuGang(vector<int> deviceIds){
	for(auto i:deviceIds){
		m_hackRfs.push_back(new HackRfGpu(i));
	}
	hipMalloc(&computeBuf,BUFLEN*sizeof(thrust::complex<float>)); 
}

HackRfGpuGang::~HackRfGpuGang(){
	for(auto hrf:m_hackRfs){
		delete hrf;
	}
}

void HackRfGpuGang::start(){
	for(auto hrf:m_hackRfs){
		hrf->start();
	}
}

void HackRfGpuGang::process(){
	Int8ToComplexFloat proc;
	BufferedMessage<thrust::complex<float>> bm(BUFLEN, (thrust::complex<float>*) computeBuf);
	
	for(auto hrf:m_hackRfs){
		try {
			BufferedMessage<int8_t>& msg = hrf->consumerCheckout();
			if(msg.m_messageLength > 0) {
				//proc.process(msg.m_ptr, msg.m_messageLength, (thrust::complex<float> *)computeBuf, BUFLEN*sizeof(thrust::complex<float>));
				proc.process(msg, bm);
			} else {
				cout << "Wierd message length " << msg.m_messageLength << " in hackrf " << hrf->m_idx << endl;
				cout << "Message buffer " << hex << static_cast<void *>(msg.m_ptr)<< endl;
			}
			hrf->consumerCheckin();
		} catch (overflow_error &e){
			cout << "Got null, itb->isStopped: " << hrf->isStopped();
		}
	}
}

void HackRfGpuGang::stop(){
	for(auto hrf:m_hackRfs){
		hrf->stop();
	}
}
