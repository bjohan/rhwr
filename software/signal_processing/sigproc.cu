#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <chrono>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "thread_stuff.hpp"
//#include "hackrf_gpu.hpp"
//#include "hackrf_gpu_gang.hpp"
#include "hackrf_thread.hpp"
#include "my_tcp_server.hpp"
#define BUFLEN 262144

using namespace std;

__global__ void VecAdd(float* A, float* B, float* C, int N){
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	if(i < N) C[i] = A[i]+B[i];
}

double getTime(){

	return(chrono::duration_cast<chrono::milliseconds>(chrono::system_clock::now().time_since_epoch()).count())/1000.0;
}
int main(int argc, char *argv[]){
	int N = 1024;
	size_t size = N*sizeof(float);
	float* h_A = (float*) malloc(size);
	float* h_B = (float*) malloc(size);
	float* h_C = (float*) malloc(size);
	double t0;
	for(int i = 0 ; i < N ; i++){
		h_A[i] = 1;
		h_B[i] = 3;
	}
	MyTcpServer srv(7000);
	srv.start();
	float* d_A;
	hipMalloc(&d_A, size);
	float* d_B;
	hipMalloc(&d_B, size);
	float* d_C;
	hipMalloc(&d_C, size);
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock -1)/threadsPerBlock;
	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


	vector<int> hackRfDeviceIndex{0, 1, 2, 3};
	HackRfThread hrt(hackRfDeviceIndex);
	hrt.start();
	//HackRfGpuGang hrg(hackRfDeviceIndex);
	//hrg.start();

	t0 =getTime();
	while(getTime()-t0 < 3){
		//hrg.process();
	}
	hrt.stop();
	cout << "exitied loop" << endl;
	//hrg.stop();
	srv.stop();
	srv.join();

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);
	free(h_C);
	return 0;
}
