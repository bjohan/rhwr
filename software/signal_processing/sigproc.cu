#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <chrono>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "thread_stuff.hpp"
#include "hackrf_gpu.hpp"
#include "hackrf_gpu_gang.hpp"
#include <matplotlibcpp.h>
#define BUFLEN 262144

using namespace std;
namespace plt = matplotlibcpp;

__global__ void VecAdd(float* A, float* B, float* C, int N){
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	if(i < N) C[i] = A[i]+B[i];
}

double getTime(){

	return(chrono::duration_cast<chrono::milliseconds>(chrono::system_clock::now().time_since_epoch()).count())/1000.0;
}
int main(int argc, char *argv[]){
	vector<double> yqplot(1024);
	vector<double> yiplot(1024);
	vector<double> absplot(1024);
	vector<double> argplot(1024);
	vector<double> freqplot(1024);
	int N = 1024;
	//float *tbuf;
	//int8_t buf[BUFLEN];
	size_t size = N*sizeof(float);
	float* h_A = (float*) malloc(size);
	float* h_B = (float*) malloc(size);
	float* h_C = (float*) malloc(size);
	double t0;
	for(int i = 0 ; i < N ; i++){
		h_A[i] = 1;
		h_B[i] = 3;
	}

	float* d_A;
	hipMalloc(&d_A, size);
	float* d_B;
	hipMalloc(&d_B, size);
	float* d_C;
	hipMalloc(&d_C, size);
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock -1)/threadsPerBlock;
	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


	vector<int> hackRfDeviceIndex{0, 1, 2, 3};
	HackRfGpuGang hrg(hackRfDeviceIndex);
	//int numHackrf=1;

	//HackRfGpu* hrfl[4];

	//for(int i = 0 ; i < numHackrf ; i++){
	//	hrfl[i] = new HackRfGpu(3);
	//}

	//for(int i = 0 ; i < numHackrf ; i++){
	//	hrfl[i]->start();
	//}
	hrg.start();

	t0 =getTime();
	while(getTime()-t0 < 1){
		//this_thread::sleep_for(chrono::milliseconds(2));
		hrg.process();
		//for(int i = 0 ; i < numHackrf ; i++){
		//	tbuf=hrfl[i]->m_itb->consumerCheckout();
		//	if(tbuf!=NULL){
		//		//cout << "reading" << tbuf << endl;
		//		if(i==0)
		//		hipMemcpy(buf, tbuf, BUFLEN, hipMemcpyDeviceToHost);
		//		hrfl[i]->m_itb->consumerCheckin();
		//	}
		//}
	}
	cout << "exitied loop" << endl;
	/*for(int i = 0 ; i < 1024 ; i++){
		yqplot[i] = (int) buf[i*2];
		yiplot[i] = (int) buf[i*2+1];
		absplot[i] = sqrt(yqplot[i]*yqplot[i]+yiplot[i]*yiplot[i]);
		argplot[i] = atan2(yiplot[i], yqplot[i]);
		if(i == 0){
			freqplot[0] = 0;
		} else {
			freqplot[i] = argplot[i]-argplot[i-1];
		}
	}*/
	/*for(int i = 0 ; i < numHackrf ; i++){
		hrfl[i]->stop();
	}*/
	hrg.stop();

	//for(int i = 0 ; i < numHackrf ; i++){
	//	delete hrfl[i];
	//}
	/*cout << "plotting" << endl;
	plt::plot(yqplot);
	plt::plot(yiplot);
	plt::plot(absplot);*/
	//plt::plot(argplot);
	//plt::plot(freqplot);
	//plt::plot(yqplot, yiplot);
	/*plt::grid(true);
	plt::show();*/

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	//for(int i = 0 ; i < N ; i++) cout << "C["<< i <<"] is "<< h_C[i] << endl;
	free(h_A);
	free(h_B);
	free(h_C);
	return 0;
}
