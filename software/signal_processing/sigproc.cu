#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <chrono>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "thread_stuff.hpp"
#include "hackrf_gpu.hpp"
#include <matplotlibcpp.h>
#define BUFLEN 262144

using namespace std;
namespace plt = matplotlibcpp;

__global__ void VecAdd(float* A, float* B, float* C, int N){
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	if(i < N) C[i] = A[i]+B[i];
}

double getTime(){

	return(chrono::duration_cast<chrono::milliseconds>(chrono::system_clock::now().time_since_epoch()).count())/1000.0;
}
int main(int argc, char *argv[]){
	vector<double> yplot(1024);
	int N = 1024;
	float *tbuf;
	int8_t buf[BUFLEN];
	size_t size = N*sizeof(float);
	float* h_A = (float*) malloc(size);
	float* h_B = (float*) malloc(size);
	float* h_C = (float*) malloc(size);
	double t0;
	for(int i = 0 ; i < N ; i++){
		h_A[i] = 1;
		h_B[i] = 3;
	}

	float* d_A;
	hipMalloc(&d_A, size);
	float* d_B;
	hipMalloc(&d_B, size);
	float* d_C;
	hipMalloc(&d_C, size);
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock -1)/threadsPerBlock;
	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


	int numHackrf=1;
	HackRfGpu* hrfl[4];

	for(int i = 0 ; i < numHackrf ; i++){
		hrfl[i] = new HackRfGpu(3);
	}

	for(int i = 0 ; i < numHackrf ; i++){
		hrfl[i]->start();
	}

	t0 =getTime();
	while(getTime()-t0 < 2){
		//this_thread::sleep_for(chrono::milliseconds(2));
		for(int i = 0 ; i < numHackrf ; i++){
			tbuf=hrfl[i]->m_itb->consumerCheckout();
			if(tbuf!=NULL){
				//cout << "reading" << tbuf << endl;
				if(i==0)
				hipMemcpy(buf, tbuf, BUFLEN, hipMemcpyDeviceToHost);
				hrfl[i]->m_itb->consumerCheckin();
			}
		}
	}
	for(int i = 0 ; i < 1024 ; i++){
		yplot[i] = (int) buf[i*2];
	}
	for(int i = 0 ; i < numHackrf ; i++){
		hrfl[i]->stop();
	}

	for(int i = 0 ; i < numHackrf ; i++){
		delete hrfl[i];
	}
	cout << "plotting" << endl;
	plt::plot(yplot);
	plt::grid(true);
	plt::show();

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	//for(int i = 0 ; i < N ; i++) cout << "C["<< i <<"] is "<< h_C[i] << endl;
	free(h_A);
	free(h_B);
	free(h_C);
	return 0;
}
