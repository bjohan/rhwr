#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <chrono>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "thread_stuff.hpp"
#include "hackrf_gpu.hpp"
#include "hackrf_gpu_gang.hpp"
#include "my_tcp_server.hpp"
#include <matplotlibcpp.h>
#define BUFLEN 262144

using namespace std;
namespace plt = matplotlibcpp;

__global__ void VecAdd(float* A, float* B, float* C, int N){
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	if(i < N) C[i] = A[i]+B[i];
}

double getTime(){

	return(chrono::duration_cast<chrono::milliseconds>(chrono::system_clock::now().time_since_epoch()).count())/1000.0;
}
int main(int argc, char *argv[]){
	vector<double> yqplot(1024);
	vector<double> yiplot(1024);
	vector<double> absplot(1024);
	vector<double> argplot(1024);
	vector<double> freqplot(1024);
	int N = 1024;
	size_t size = N*sizeof(float);
	float* h_A = (float*) malloc(size);
	float* h_B = (float*) malloc(size);
	float* h_C = (float*) malloc(size);
	double t0;
	for(int i = 0 ; i < N ; i++){
		h_A[i] = 1;
		h_B[i] = 3;
	}
	MyTcpServer srv(7000);
	srv.start();
	float* d_A;
	hipMalloc(&d_A, size);
	float* d_B;
	hipMalloc(&d_B, size);
	float* d_C;
	hipMalloc(&d_C, size);
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock -1)/threadsPerBlock;
	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


	vector<int> hackRfDeviceIndex{0, 1, 2, 3};
	HackRfGpuGang hrg(hackRfDeviceIndex);
	hrg.start();

	t0 =getTime();
	while(getTime()-t0 < 30){
		hrg.process();
	}
	cout << "exitied loop" << endl;
	hrg.stop();
	srv.stop();
	srv.join();

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);
	free(h_C);
	return 0;
}
