#include "buffered_message.hpp"
#include <thrust/complex.h>
using namespace std;
/*template<typename T>
BufferedMessage<T>::BufferedMessage(int buflen, T* bufPtr){
	m_buflen = buflen;
	m_ptr = bufPtr;
}


template<typename T>
BufferedMessage<T>::BufferedMessage(){
	m_buflen = 0;
	m_ptr = NULL;
}

template<typename T>
void BufferedMessage<T>::allocate(int msgsize){
	m_buflen = 0;
	m_ptr = NULL;
}

template<typename T>
void BufferedMessage<T>::setBufferLength(int buflen){
	m_buflen = buflen;
}

template<typename T>
void BufferedMessage<T>::toStream(ostream &os){
	os << "this address " << hex << static_cast<void*>(this) << " m_ptr " << hex << static_cast<void*>(m_ptr) << " m_buflen " << m_buflen << " m_messageLength " << m_messageLength;
}

template<typename T>
int BufferedMessage<T>::len(){
	return m_buflen;
}*/
/*
template<typename T>
BufferedMessageDevice<T>::BufferedMessageDevice(int buflen){
	allocate(buflen);
}

template<typename T>
void BufferedMessageDevice<T>::allocate(int buflen){
	hipMalloc(&(this->m_ptr), buflen*sizeof(T));
	BufferedMessage<T>::m_buflen = buflen;
}

template<typename T>
BufferedMessageDevice<T>::~BufferedMessageDevice(){
	hipFree(BufferedMessage<T>::m_ptr);
	BufferedMessage<T>::m_ptr = NULL;
	BufferedMessage<T>::m_buflen = 0;
}


template<typename T>
BufferedMessageHost<T>::BufferedMessageHost(int buflen){
	allocate(buflen);
}

template<typename T>
void BufferedMessageHost<T>::allocate(int buflen){
	BufferedMessage<T>::m_ptr = new T[buflen];
	BufferedMessage<T>::m_buflen = buflen;
}

template<typename T>
BufferedMessageHost<T>::~BufferedMessageHost(){
	delete BufferedMessage<T>::m_ptr;
	BufferedMessage<T>::m_ptr = NULL;
	BufferedMessage<T>::m_buflen = 0;
}



template class BufferedMessage<float>;
template class BufferedMessage<int8_t>;
template class BufferedMessage<char>;
template class BufferedMessage<thrust::complex<float>>;

template class BufferedMessageHost<float>;
template class BufferedMessageHost<int8_t>;
template class BufferedMessageHost<char>;
template class BufferedMessageHost<thrust::complex<float>>;

template class BufferedMessageDevice<float>;
template class BufferedMessageDevice<int8_t>;
template class BufferedMessageDevice<char>;
template class BufferedMessageDevice<thrust::complex<float>>;
*/
