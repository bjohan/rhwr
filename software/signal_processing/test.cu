#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <chrono>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <assert.h>
#include <libhackrf/hackrf.h>
#include <hip/hip_runtime.h>
#define BUFLEN 262144
using namespace std;


class MyHackRf{
	private:
		static int rx_callback(hackrf_transfer* transfer);
		static int refCount;
		static hackrf_device_list_t* devs;

	public:
		float *gpuBuf;
		bool running;
		hackrf_device *dev = NULL;
		MyHackRf(int index);
		~MyHackRf();
		void start();
		void stop();

};

int MyHackRf::refCount=0;
hackrf_device_list_t* MyHackRf::devs=NULL;

MyHackRf::MyHackRf(int index){
	int status;
	if(refCount == 0){
		cout << "First hackrf object created, calling hackrf_init() and creating device list" << endl;
		hackrf_init();
		devs = hackrf_device_list();
	}
	refCount++;
	status = hackrf_device_list_open(devs, index, &dev);
	if(status) cout << "Failed to open hackrf index: " << index << " status: " << status << endl;

	hipMalloc((void**) &gpuBuf, sizeof(float)*BUFLEN);
	running = false;

}

int MyHackRf::rx_callback(hackrf_transfer* transfer){
	hipMemcpy(transfer->buffer, transfer->rx_ctx, transfer->valid_length, hipMemcpyHostToDevice);
	return 0;
}

void MyHackRf::start(){
	if(dev != NULL){
		hackrf_start_rx(dev, rx_callback, (void *) gpuBuf);
		running = true;
	} else {
		cout << "hackrf device is not open" << endl;
	}
}

void MyHackRf::stop(){
	if(running){
		hackrf_stop_rx(dev);
		running = false;
	}
}

MyHackRf::~MyHackRf(){
	stop();
	hackrf_close(dev);
	hipFree(gpuBuf);
	refCount--;
	if(refCount == 0){
		cout << "Last hackrf object is beeing destroyed, freeing hackrf device list and deiniting library" << endl;
		hackrf_device_list_free(devs);
	       	hackrf_exit();
	}
}
__global__ void VecAdd(float* A, float* B, float* C, int N){
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	if(i < N) C[i] = A[i]+B[i];
}

int rx_callback(hackrf_transfer* transfer) {
	hipMemcpy(transfer->buffer, transfer->rx_ctx, transfer->valid_length, hipMemcpyHostToDevice);
	//printf("buffer length %d, data length %d\n", transfer->buffer_length, transfer->valid_length);
	return 0;
}

int main(int argc, char *argv[]){
	cout << "hejsan" << endl;
	int N = 1024;
	size_t size = N*sizeof(float);
	float* h_A = (float*) malloc(size);
	float* h_B = (float*) malloc(size);
	float* h_C = (float*) malloc(size);
	for(int i = 0 ; i < N ; i++){
		h_A[i] = 1;
		h_B[i] = 3;
	}

	float* d_A;
	hipMalloc(&d_A, size);
	float* d_B;
	hipMalloc(&d_B, size);
	float* d_C;
	hipMalloc(&d_C, size);
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock -1)/threadsPerBlock;
	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


	int numHackrf=4;
	MyHackRf* hrfl[4];

	for(int i = 0 ; i < numHackrf ; i++){
		hrfl[i] = new MyHackRf(i);
	}

	for(int i = 0 ; i < numHackrf ; i++){
		hrfl[i]->start();
	}

	this_thread::sleep_for(chrono::seconds(30));

	for(int i = 0 ; i < numHackrf ; i++){
		hrfl[i]->stop();
	}

	for(int i = 0 ; i < numHackrf ; i++){
		delete hrfl[i];
	}



	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	//for(int i = 0 ; i < N ; i++) cout << "C["<< i <<"] is "<< h_C[i] << endl;
	free(h_A);
	free(h_B);
	free(h_C);
	return 0;
}
