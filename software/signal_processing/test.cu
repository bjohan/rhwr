#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <chrono>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "thread_stuff.hpp"
#include "hackrf_gpu.hpp"

#define BUFLEN 262144

using namespace std;


__global__ void VecAdd(float* A, float* B, float* C, int N){
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	if(i < N) C[i] = A[i]+B[i];
}


int main(int argc, char *argv[]){
	int N = 1024;
	size_t size = N*sizeof(float);
	float* h_A = (float*) malloc(size);
	float* h_B = (float*) malloc(size);
	float* h_C = (float*) malloc(size);
	for(int i = 0 ; i < N ; i++){
		h_A[i] = 1;
		h_B[i] = 3;
	}

	float* d_A;
	hipMalloc(&d_A, size);
	float* d_B;
	hipMalloc(&d_B, size);
	float* d_C;
	hipMalloc(&d_C, size);
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock -1)/threadsPerBlock;
	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


	int numHackrf=4;
	HackRfGpu* hrfl[4];

	for(int i = 0 ; i < numHackrf ; i++){
		hrfl[i] = new HackRfGpu(i);
	}

	for(int i = 0 ; i < numHackrf ; i++){
		hrfl[i]->start();
	}

	this_thread::sleep_for(chrono::seconds(30));

	for(int i = 0 ; i < numHackrf ; i++){
		hrfl[i]->stop();
	}

	for(int i = 0 ; i < numHackrf ; i++){
		delete hrfl[i];
	}



	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	//for(int i = 0 ; i < N ; i++) cout << "C["<< i <<"] is "<< h_C[i] << endl;
	free(h_A);
	free(h_B);
	free(h_C);
	return 0;
}
