#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <chrono>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "thread_stuff.hpp"
#include "hackrf_wrapper.hpp"

#define BUFLEN 262144

using namespace std;

/*template < class T>
class InterThreadBuffer{
	private:
		T m_buffs[];
		int m_producerPos=0;
		int m_consumerPos=0;
		int m_sz = 0;
		mutex m_mutex;

	public:
		InterThreadBuffer(int sz);
		~InterThreadBuffer();
		T producerCheckout();
		void producerCheckin(T);
		T consumerCheckout();
		void consumerCheckin();
		T getBufferUnsafe(int i);
};

template < class T>
InterThreadBuffer<T>::InterThreadBuffer(int sz){
	m_buffs = new T[sz];
	m_sz = sz;
	for(int i = 0 ; i < sz ; i++) m_buffs[i]=NULL;
}

template < class T>
InterThreadBuffer<T>::~InterThreadBuffer(){
	delete[] m_buffs;
}

template <class T>
T InterThreadBuffer<T>::producerCheckout(){
	T result;
	m_mutex.lock();
	int next = (m_producerPos+1)%m_sz;
	if(next == m_consumerPos){
		m_mutex.unlock();
		return NULL;
	}
	result = m_buffs[m_producerPos];
	m_mutex.unlock();
	return result;
}

template <class T>
void InterThreadBuffer<T>::producerCheckin(T p){
	m_mutex.lock();
	int next = (m_consumerPos+1)%m_sz;
	if(next == m_producerPos){
		m_mutex.unlock();
	       	throw "Error, try to commit to full buffer";
	}
	m_buffs[m_producerPos] = p;
	m_producerPos = next;
	m_mutex.unlock();
}

template <class T>
T InterThreadBuffer<T>::consumerCheckout(){
	T result;
	m_mutex.lock();
	if(m_consumerPos != m_producerPos){
		m_mutex.unlock();
	       return NULL;
	}
	result = m_buffs[m_consumerPos];
	m_mutex.unlock();
	return result;
}

template <class T>
void InterThreadBuffer<T>::consumerCheckin(){
	m_mutex.lock();
	int next = (m_consumerPos+1)%m_sz;
	m_consumerPos = next;
	m_mutex.unlock();
}

template <class T>
T InterThreadBuffer<T>::getBufferUnsafe(int i){
	return m_buffs[i];
}

class MyHackRf{
	private:
		static int rx_callback(hackrf_transfer* transfer);
		static int refCount;
		static hackrf_device_list_t* devs;

	public:
		//float *gpuBuf;
		bool running;
		hackrf_device *dev = NULL;
		MyHackRf(int index);
		virtual int myRxCallback(hackrf_transfer *transfer);
		~MyHackRf();
		void start();
		void stop();

};

int MyHackRf::refCount=0;
hackrf_device_list_t* MyHackRf::devs=NULL;

MyHackRf::MyHackRf(int index){
	int status;
	if(refCount == 0){
		cout << "First hackrf object created, calling hackrf_init() and creating device list" << endl;
		hackrf_init();
		devs = hackrf_device_list();
	}
	refCount++;
	status = hackrf_device_list_open(devs, index, &dev);
	if(status) cout << "Failed to open hackrf index: " << index << " status: " << status << endl;

	//hipMalloc((void**) &gpuBuf, sizeof(float)*BUFLEN);
	running = false;

}

int MyHackRf::myRxCallback(hackrf_transfer* transfer){
	//hipMemcpy(transfer->buffer, gpuBuf, transfer->valid_length, hipMemcpyHostToDevice);
	return 0;
}

int MyHackRf::rx_callback(hackrf_transfer* transfer){
	return ((MyHackRf *)transfer->rx_ctx)->myRxCallback(transfer);
	//hipMemcpy(transfer->buffer, transfer->rx_ctx, transfer->valid_length, hipMemcpyHostToDevice);
	//return 0;
}

void MyHackRf::start(){
	if(dev != NULL){
		hackrf_start_rx(dev, rx_callback, (void *) this);
		running = true;
	} else {
		cout << "hackrf device is not open" << endl;
	}
}

void MyHackRf::stop(){
	if(running){
		hackrf_stop_rx(dev);
		running = false;
	}
}

MyHackRf::~MyHackRf(){
	stop();
	hackrf_close(dev);
	//hipFree(gpuBuf);
	refCount--;
	if(refCount == 0){
		cout << "Last hackrf object is beeing destroyed, freeing hackrf device list and deiniting library" << endl;
		hackrf_device_list_free(devs);
	       	hackrf_exit();
	}
}
*/



__global__ void VecAdd(float* A, float* B, float* C, int N){
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	if(i < N) C[i] = A[i]+B[i];
}


int main(int argc, char *argv[]){
	int N = 1024;
	size_t size = N*sizeof(float);
	float* h_A = (float*) malloc(size);
	float* h_B = (float*) malloc(size);
	float* h_C = (float*) malloc(size);
	for(int i = 0 ; i < N ; i++){
		h_A[i] = 1;
		h_B[i] = 3;
	}

	float* d_A;
	hipMalloc(&d_A, size);
	float* d_B;
	hipMalloc(&d_B, size);
	float* d_C;
	hipMalloc(&d_C, size);
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock -1)/threadsPerBlock;
	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


	int numHackrf=4;
	MyHackRf* hrfl[4];

	for(int i = 0 ; i < numHackrf ; i++){
		hrfl[i] = new MyHackRf(i);
	}

	for(int i = 0 ; i < numHackrf ; i++){
		hrfl[i]->start();
	}

	this_thread::sleep_for(chrono::seconds(30));

	for(int i = 0 ; i < numHackrf ; i++){
		hrfl[i]->stop();
	}

	for(int i = 0 ; i < numHackrf ; i++){
		delete hrfl[i];
	}



	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	//for(int i = 0 ; i < N ; i++) cout << "C["<< i <<"] is "<< h_C[i] << endl;
	free(h_A);
	free(h_B);
	free(h_C);
	return 0;
}
