#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <chrono>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "thread_stuff.hpp"
#include "hackrf_gpu.hpp"

#define BUFLEN 262144

using namespace std;


__global__ void VecAdd(float* A, float* B, float* C, int N){
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	if(i < N) C[i] = A[i]+B[i];
}

double getTime(){

	return(chrono::duration_cast<chrono::milliseconds>(chrono::system_clock::now().time_since_epoch()).count())/1000.0;
}
int main(int argc, char *argv[]){
	int N = 1024;
	float *tbuf;
	size_t size = N*sizeof(float);
	float* h_A = (float*) malloc(size);
	float* h_B = (float*) malloc(size);
	float* h_C = (float*) malloc(size);
	double t0;
	for(int i = 0 ; i < N ; i++){
		h_A[i] = 1;
		h_B[i] = 3;
	}

	float* d_A;
	hipMalloc(&d_A, size);
	float* d_B;
	hipMalloc(&d_B, size);
	float* d_C;
	hipMalloc(&d_C, size);
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock -1)/threadsPerBlock;
	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


	int numHackrf=4;
	HackRfGpu* hrfl[4];

	for(int i = 0 ; i < numHackrf ; i++){
		hrfl[i] = new HackRfGpu(i);
	}

	for(int i = 0 ; i < numHackrf ; i++){
		hrfl[i]->start();
	}

	t0 =getTime();
	while(getTime()-t0 < 10){
		this_thread::sleep_for(chrono::milliseconds(2));
		for(int i = 0 ; i < numHackrf ; i++){
			tbuf=hrfl[i]->m_itb->consumerCheckout();
			if(tbuf!=NULL)
				hrfl[i]->m_itb->consumerCheckin();
		}
	}
	for(int i = 0 ; i < numHackrf ; i++){
		hrfl[i]->stop();
	}

	for(int i = 0 ; i < numHackrf ; i++){
		delete hrfl[i];
	}



	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	//for(int i = 0 ; i < N ; i++) cout << "C["<< i <<"] is "<< h_C[i] << endl;
	free(h_A);
	free(h_B);
	free(h_C);
	return 0;
}
