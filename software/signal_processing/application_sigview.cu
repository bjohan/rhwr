#include <iostream>
#include <thread>
#include <chrono>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <assert.h>
#include "simple_tcp.hpp"
#include <readline/readline.h>
#include <readline/history.h>

using namespace std;


double getTime(){

	return(chrono::duration_cast<chrono::milliseconds>(chrono::system_clock::now().time_since_epoch()).count())/1000.0;
}
int main(int argc, char *argv[]){
	TcpClient("localhost", 7000);	
	t0 =getTime();
	char *buf;
	while(true){
		buf = readline(">>");
		if(buf){
			if(strlen(buf)){
				add_history(buf);
				cout << "Command entered" << buf << " at run time " << getTime()-t0 << endl;
			}
		}
		if(strcmp(buf, "quit")==0) break;
		free(buf);
		//hrg.process();
	}
	hrt.stop();
	cout << "exitied loop" << endl;
	//hrg.stop();
	srv.stop();
	srv.join();

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);
	free(h_C);
	return 0;
}
