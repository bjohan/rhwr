#include "hip/hip_runtime.h"
//#include <thrust/complex.h>
#include "int8_to_complex_float.hpp"
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;


__global__ void IntToFloat(int8_t *in, thrust::complex<float> *out, int N){
	int i = blockDim.x*blockIdx.x+threadIdx.x;

	if(i < N) out[i] = thrust::complex<float>(in[2*i], in[2*i+1]);
}




Int8ToComplexFloat::Int8ToComplexFloat()
{
}

//void Int8ToComplexFloat::process(int8_t *in, size_t sizeIndata, thrust::complex<float> *out, size_t maxSizeOutData){
void Int8ToComplexFloat::process(BufferedMessage<int8_t> &in, BufferedMessage<thrust::complex<float>> &out){
	int outDataElems = in.m_messageLength/2;//sizeIndata/2;
	int inDataElems = in.m_messageLength/2; //sizeIndata/2;
	if(outDataElems > out.len()){
		cout << "Out data does not fit in designated array. indataElems " << inDataElems <<  " outDataElems " << outDataElems << " maxOutDataElems " << out.len() << endl;
	}
        	
	int threadsPerBlock = 32;
	int blocksPerGrid = (inDataElems + threadsPerBlock -1)/threadsPerBlock;
	IntToFloat<<<blocksPerGrid, threadsPerBlock>>>(in.m_ptr, out.m_ptr, inDataElems);
	//cout << "processed " << inDataElems << " samples" << endl;	
}
