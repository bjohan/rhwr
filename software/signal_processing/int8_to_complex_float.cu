#include "hip/hip_runtime.h"
//#include <thrust/complex.h>
#include "int8_to_complex_float.hpp"
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;


__global__ void IntToFloat(int8_t *in, thrust::complex<float> *out, int N){
	int i = blockDim.x*blockIdx.x+threadIdx.x;

	if(i < N) out[i] = thrust::complex<float>(in[2*i], in[2*i+1]);
}




Int8ToComplexFloat::Int8ToComplexFloat()
{
}

void Int8ToComplexFloat::process(int8_t *in, size_t sizeIndata, thrust::complex<float> *out, size_t maxSizeOutData){
	int outDataElems = sizeIndata/2;
	int inDataElems = sizeIndata/2;
	size_t outSize = outDataElems*sizeof(thrust::complex<float>);
	if(outSize > maxSizeOutData){
		cout << "Out data does not fit in designated array" << endl;
	}
        	
	int threadsPerBlock = 32;
	int blocksPerGrid = (inDataElems + threadsPerBlock -1)/threadsPerBlock;
	IntToFloat<<<blocksPerGrid, threadsPerBlock>>>(in, out, inDataElems);
	//cout << "processed " << inDataElems << " samples" << endl;	
}
