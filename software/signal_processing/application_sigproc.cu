#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "thread_stuff.hpp"
//#include "hackrf_gpu.hpp"
//#include "hackrf_gpu_gang.hpp"
#include "hackrf_thread.hpp"
#include "my_tcp_server.hpp"
#define BUFLEN 262144
#include <readline/readline.h>
#include <readline/history.h>
#include "aux_util.hpp"
#include "signal_server.hpp" 


using namespace std;

__global__ void VecAdd(float* A, float* B, float* C, int N){
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	if(i < N) C[i] = A[i]+B[i];
}


int main(int argc, char *argv[]){
	int N = 1024;
	size_t size = N*sizeof(float);
	float* h_A = (float*) malloc(size);
	float* h_B = (float*) malloc(size);
	float* h_C = (float*) malloc(size);
	double t0;
	for(int i = 0 ; i < N ; i++){
		h_A[i] = 1;
		h_B[i] = 3;
	}
	MyTcpServer srv(7000);
	srv.start();
	SignalServer sigsrv(srv.pub);
	sigsrv.start();
	float* d_A;
	hipMalloc(&d_A, size);
	float* d_B;
	hipMalloc(&d_B, size);
	float* d_C;
	hipMalloc(&d_C, size);
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock -1)/threadsPerBlock;
	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


	vector<int> hackRfDeviceIndex{0, 1, 2, 3};
	HackRfThread hrt(hackRfDeviceIndex);
	hrt.start();
	//HackRfGpuGang hrg(hackRfDeviceIndex);
	//hrg.start();

	t0 =getTime();
	char *buf;
	while(true){
		buf = readline(">> ");
		if(buf){
			if(strlen(buf)){
				add_history(buf);
				cout << "Command entered" << buf << " at run time " << getTime()-t0 << endl;
			}
		}
		if(strcmp(buf, "quit")==0) break;
		free(buf);
		//hrg.process();
	}
	hrt.stop();
	cout << "exitied loop" << endl;
	//hrg.stop();


	sigsrv.stop();
	cout << "joining signal server" << endl;
	sigsrv.join();


	srv.stop();
	cout << "joining server thread" << endl;
	srv.join();

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);
	free(h_C);
	cout << "return from main";
	return 0;
}
