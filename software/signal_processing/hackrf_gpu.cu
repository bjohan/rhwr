#include "hackrf_gpu.hpp"
#include <stdexcept>
#include <iostream>
#define BUFLEN 262144
using namespace std;

HackRfGpu::HackRfGpu(int index) : MyHackRf(index), m_itb(8){
	for(int i = 0 ; i < m_itb.getSize() ; i++){
		BufferedMessage<int8_t> &msg = m_itb.getBufferIndexUnsafe(i);
		hipMalloc(&(msg.m_ptr), BUFLEN);
		msg.setBufferLength(BUFLEN);
		msg.m_messageLength = 0;
	}
	m_success = 0;
	m_fail = 0;
}

HackRfGpu::~HackRfGpu(){
	stop();
	for(int i = 0 ; i < m_itb.getSize() ; i++){
		auto &msg = m_itb.getBufferIndexUnsafe(i);
		hipFree(msg.m_ptr);
		msg.setBufferLength(0);
	}
	cout << "Hackrf " << m_idx << dec << " was destroyed. Result; Success: " << m_success << " Fails: " << m_fail << endl;
	
}

void HackRfGpu::myStop(){
	cout << "My stop called" << endl;
	m_itb.producerStop();
}

int HackRfGpu::myRxCallback(hackrf_transfer* xfer){
	try {
		if(xfer->valid_length > 0){
			auto &msg = m_itb.producerCheckout();
			hipMemcpy(msg.m_ptr, xfer->buffer, xfer->valid_length, hipMemcpyHostToDevice);
			msg.m_messageLength = xfer->valid_length;
			m_success+=xfer->valid_length/2;
			m_itb.producerCheckin();
		} else {
			cout << "Valid length " << xfer->valid_length << endl;
		}
	} catch (overflow_error &e) {
		m_fail+=xfer->valid_length/2;
	}
	return 0;
}

