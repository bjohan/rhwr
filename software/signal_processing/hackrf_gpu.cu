#include "hackrf_gpu.hpp"
#include <stdexcept>
#include <iostream>
#include "aux_util.hpp"
#define BUFLEN 262144
using namespace std;

HackRfGpu::HackRfGpu(int index) : MyHackRf(index), DataSource(8, BUFLEN){
	/*for(int i = 0 ; i < m_itb.getSize() ; i++){
		BufferedMessage<int8_t> &msg = m_itb.getBufferIndexUnsafe(i);
		msg.allocate(BUFLEN);
		//hipMalloc(&(msg.m_ptr), BUFLEN);
		//msg.setBufferLength(BUFLEN);
		//msg.m_messageLength = 0;
	}*/
	m_tStart = getTime();
	cout << "Starting hackrf at " << m_tStart << endl;
	m_success = 0;
	m_fail = 0;
}

HackRfGpu::~HackRfGpu(){
	stop();
	/*for(int i = 0 ; i < m_itb.getSize() ; i++){
		auto &msg = m_itb.getBufferIndexUnsafe(i);
		hipFree(msg.m_ptr);
		msg.setBufferLength(0);
	}*/
	cout << "Hackrf " << m_idx << dec << " was destroyed. Result; Success: " << m_success << " Fails: " << m_fail << endl;
	cout << "ran for " << m_runTime << " with a rate of " << m_success/m_runTime<< endl;
	
}

void HackRfGpu::myStop(){
	double stopTime = getTime();
	m_runTime = stopTime-m_tStart;
	cout << "My stop called a  " << stopTime << endl;
	producerStop();
}

int HackRfGpu::myRxCallback(hackrf_transfer* xfer){
	try {
		if(xfer->valid_length > 0){
			auto &msg = producerCheckout();
			hipMemcpy(msg.m_ptr, xfer->buffer, xfer->valid_length, hipMemcpyHostToDevice);
			msg.m_messageLength = xfer->valid_length;
			m_success+=xfer->valid_length/2;
			producerCheckin();
		} else {
			cout << "Valid length " << xfer->valid_length << endl;
		}
	} catch (overflow_error &e) {
		m_fail+=xfer->valid_length/2;
	}
	return 0;
}

