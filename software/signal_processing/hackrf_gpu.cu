#include "hackrf_gpu.hpp"
#include <iostream>
#define BUFLEN 262144
using namespace std;

HackRfGpu::HackRfGpu(int index) : MyHackRf(index){
	float **p;
	m_itb = new InterThreadBuffer<float *>(8);
	for(int i = 0 ; i < m_itb->getSize() ; i++){
		p = m_itb->getBufferIndexUnsafe(i);
		hipMalloc(p, BUFLEN*sizeof(float));
	}
	m_success = 0;
	m_fail = 0;
}

HackRfGpu::~HackRfGpu(){
	float **p;
	stop();
	for(int i = 0 ; i < m_itb->getSize() ; i++){
		p = m_itb->getBufferIndexUnsafe(i);
		hipFree(*p);
	}
	delete m_itb;
	cout << "Hackrf " << m_idx << " Success: " << m_success << " Fails: " << m_fail << endl;
	
}

int HackRfGpu::myRxCallback(hackrf_transfer* xfer){
	float *mem = m_itb->producerCheckout();
	if(mem != NULL){
		hipMemcpy(xfer->buffer, mem, xfer->valid_length, hipMemcpyHostToDevice);
		m_success+=xfer->valid_length/2;
		m_itb->producerCheckin(mem);
	} else {
		m_fail+=xfer->valid_length/2;
	}
	return 0;
}

