#include "hackrf_gpu.hpp"
#include <iostream>
#define BUFLEN 262144
using namespace std;

HackRfGpu::HackRfGpu(int index) : MyHackRf(index){
	float **p;
	m_itb = new InterThreadBuffer<float *>(8);
	for(int i = 0 ; i < m_itb->getSize() ; i++){
		p = m_itb->getBufferIndexUnsafe(i);
		hipMalloc(p, BUFLEN*sizeof(float));
		//cout << "Allocated " << *p << endl;
	}
	m_success = 0;
	m_fail = 0;
}

HackRfGpu::~HackRfGpu(){
	float **p;
	stop();
	for(int i = 0 ; i < m_itb->getSize() ; i++){
		p = m_itb->getBufferIndexUnsafe(i);
		//cout << "freed " << *p << endl;
		hipFree(p);
	}
	delete m_itb;
	cout << "Hackrf " << m_idx << " Success: " << m_success << " Fails: " << m_fail << endl;
	
}

void HackRfGpu::myStop(){
	cout << "My stop called" << endl;
	m_itb->producerStop();
}

int HackRfGpu::myRxCallback(hackrf_transfer* xfer){
	float *mem = m_itb->producerCheckout();
	if(mem != NULL){
		//cout << "copy to" << mem << " " << endl;
		hipMemcpy(mem, xfer->buffer, xfer->valid_length, hipMemcpyHostToDevice);
		m_success+=xfer->valid_length/2;
		m_itb->producerCheckin(mem);
	} else {
		m_fail+=xfer->valid_length/2;
	}
	return 0;
}

